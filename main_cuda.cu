#include "hip/hip_runtime.h"
#include "macro.hpp"
#include <algorithm>
#include <iostream>
#include <cstdlib>

#define double float

int RandomNumber(){return static_cast<double>(rand() % 1000);}

__global__ void sum3(double * a,
		     double * b,
		     double * c,
		     double * result,
		     unsigned size)
		     
{
  unsigned i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i < size)
    {
      result[i] = (a[i] + b[i] + c[i]);
    }
};

int main()
{
  
  unsigned size = 1e7;
  srand(0);
  double* a = new double[size];
  double* b = new double[size];
  double* c = new double[size];
  double* result = new double[size];
  
  std::generate(a, a+size, RandomNumber);
  std::generate(b, b+size, RandomNumber);
  std::generate(c, c+size, RandomNumber);

  double* ad, *bd,* cd;
  double* resultd;

  unsigned * sized;
 hipMalloc((void**) &ad, size*sizeof(double)) ;
 hipMalloc((void**) &bd, size*sizeof(double)) ;
 hipMalloc((void**) &cd, size*sizeof(double)) ;
 hipMalloc((void**) &resultd, size*sizeof(double));
 hipMalloc((void**) &sized, sizeof(unsigned)) ;

  for(int i = 0; i < 1000; ++i)
    {
      unsigned block_size = 515;
      unsigned num_blocks = (size + block_size - 1) / block_size;
      hipMemcpy(ad, a, size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(bd, b, size*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(cd, c, size*sizeof(double), hipMemcpyHostToDevice);            
      sum3<<<num_blocks, block_size>>>(ad, bd, cd, resultd, size);
      hipMemcpy(result, resultd, size*sizeof(double), hipMemcpyDeviceToHost);
    }

#ifdef PRINT
  for( int i = 0; i < size; ++i)
    {
      std::cout << a[i] << ", "<< b[i] <<"," << c[i] << "," << result[i]<< std::endl;
    }
#endif

  hipFree(ad);
  hipFree(bd);
  hipFree(cd);
  hipFree(resultd);

  delete[] a;
  delete[] b;
  delete[] c;
  delete[] result;
  
  return 0;
}
